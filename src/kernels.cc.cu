#include "hip/hip_runtime.h"
#include "kernels.h"
#include <hip/hip_complex.h>
#include "matrix.hpp"
#include "utils.hpp"
#include "n_aryGrayCodeCounter.hpp"

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
__device__ double atomicAdd(double *address, double val)
{
  unsigned long long int *address_as_ull = (unsigned long long int *)address;
  unsigned long long int old = *address_as_ull;
  unsigned long long int assumed;
  do
  {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}
#endif

__device__ void atAddComplex(hipDoubleComplex *a, hipDoubleComplex b)
{
  double *x = (double *)a;
  double *y = x + 1;
  // use atomicAdd for double variables
  atomicAdd(x, hipCreal(b));
  atomicAdd(y, hipCimag(b));
}

template <ffi::DataType T>
std::pair<int64_t, int64_t> get_dims(const ffi::Buffer<T> &buffer)
{
  auto dims = buffer.dimensions();

  if (dims.size() == 0)
  {
    return std::make_pair(0, 0);
  }
  return std::make_pair(buffer.element_count(), dims.back());
}

__global__ void PermanentKernelMatrix(Matrix<hipDoubleComplex> A, uint64_t *rows, size_t rows_size,
                                      uint64_t *cols, size_t cols_size,
                                      int *h_n_ary_limits, size_t n_ary_size, uint64_t idx_max,
                                      int64_t host_max_concurrent_warps, int sum_rows, hipDoubleComplex *result)
{
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  size_t stride = blockDim.x * gridDim.x;

  hipDoubleComplex local_result = make_hipDoubleComplex(0.0, 0.0);

  int64_t concurrency = min(host_max_concurrent_warps, static_cast<int64_t>(idx_max));

  for (uint64_t job_idx = tid; job_idx < concurrency; job_idx += stride)
  {
    int64_t work_batch = idx_max / concurrency;
    int64_t initial_offset = job_idx * work_batch;
    int64_t offset_max = (job_idx + 1) * work_batch - 1;
    if (job_idx == concurrency - 1)
    {
      offset_max = idx_max - 1;
    }

    n_aryGrayCodeCounter gcode_counter(h_n_ary_limits, n_ary_size, initial_offset);
    gcode_counter.set_offset_max(offset_max);

    int *gcode = gcode_counter.get();
    int binomial_coeff = 1;
    int minus_signs_all = 0;

    hipDoubleComplex colsum[64];
    for (size_t i = 0; i < cols_size && i < 64; i++)
    {
      colsum[i] = A(0, i);
    }

    for (size_t row = 0; row < n_ary_size; row++)
    {
      int minus_signs = gcode[row];
      int row_mult = rows[row + 1];
      for (size_t col = 0; col < cols_size && col < 64; col++)
      {
        double factor = row_mult - 2.0 * minus_signs;
        hipDoubleComplex scaled = hipCmul(A(row + 1, col),
                                        make_hipDoubleComplex(factor, 0.0));
        colsum[col] = hipCadd(colsum[col], scaled);
      }

      minus_signs_all += minus_signs;

      binomial_coeff *= binomialCoeffManual<int>(row_mult, minus_signs);
    }

    int parity = (minus_signs_all % 2 == 0) ? 1 : -1;

    hipDoubleComplex colsum_prod = make_hipDoubleComplex((double)parity, 0.0);

    for (size_t i = 0; i < cols_size && i < 64; i++)
    {
      for (size_t j = 0; j < cols[i]; j++)
      {
        colsum_prod = hipCmul(colsum_prod, colsum[i]);
      }
    }

    colsum_prod = hipCmul(colsum_prod, make_hipDoubleComplex((double)binomial_coeff, 0.0));

    local_result = hipCadd(local_result, colsum_prod);

    for (int64_t idx = initial_offset + 1; idx < offset_max + 1; idx++)
    {
      int changed_index, prev_value, value;
      if (gcode_counter.next(changed_index, prev_value, value))
      {
        break;
      }

      parity = -parity;

      int row_offset = changed_index + 1;
      hipDoubleComplex colsum_prod = make_hipDoubleComplex(parity, 0.0);
      for (size_t col_idx = 0; col_idx < cols_size; col_idx++)
      {
        if (prev_value < value)
        {
          colsum[col_idx] = hipCsub(colsum[col_idx], hipCmul(make_hipDoubleComplex(2.0, 0.0), A(row_offset, col_idx)));
        }
        else
        {
          colsum[col_idx] = hipCadd(colsum[col_idx], hipCmul(make_hipDoubleComplex(2.0, 0.0), A(row_offset, col_idx)));
        }

        for (size_t jdx = 0; jdx < cols[col_idx]; jdx++)
        {
          colsum_prod = hipCmul(colsum_prod, colsum[col_idx]);
        }
      }

      int row_mult_current = rows[changed_index + 1];
      binomial_coeff =
          value < prev_value
              ? binomial_coeff * prev_value / (row_mult_current - value)
              : binomial_coeff * (row_mult_current - prev_value) / value;

      colsum_prod = hipCmul(colsum_prod, make_hipDoubleComplex((double)binomial_coeff, 0.0));
      local_result = hipCadd(local_result, colsum_prod);
    }
  }
  double scale_factor = 1.0 / (1ULL << (sum_rows - 1));
  local_result = hipCmul(local_result, make_hipDoubleComplex(scale_factor, 0.0));

  atAddComplex(result, local_result);
}

hipError_t calculatePermanent(hipStream_t stream,
                               hipDoubleComplex *A_data, size_t n,
                               uint64_t *rows_data, size_t rows_size,
                               uint64_t *cols_data, size_t cols_size,
                               hipDoubleComplex *permanent_data)
{
  std::vector<uint64_t> h_rows(rows_size);
  hipError_t cuda_err = hipMemcpy(h_rows.data(), rows_data, rows_size * sizeof(uint64_t), hipMemcpyDeviceToHost);
  if (cuda_err != hipSuccess)
    return cuda_err;

  int sum_rows = 0;
  for (auto r : h_rows)
    sum_rows += r;

  std::vector<uint64_t> h_cols(cols_size);
  cuda_err = hipMemcpy(h_cols.data(), cols_data, cols_size * sizeof(uint64_t), hipMemcpyDeviceToHost);
  if (cuda_err != hipSuccess)
    return cuda_err;

  int sum_cols = 0;
  for (auto c : h_cols)
    sum_cols += c;

  size_t min_idx = 0;
  int minelem = 0;

  for (int i = 0; i < h_rows.size(); i++)
  {
    if (minelem == 0 || (h_rows[i] < minelem && h_rows[i] != 0))
    {
      minelem = h_rows[i];
      min_idx = i;
    }
  }

  int device;
  hipGetDevice(&device);
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, device);
  unsigned int warps_per_sm = props.warpSize > 0 ? (props.maxThreadsPerMultiProcessor / props.warpSize) : 32;
  int64_t host_max_concurrent_warps = (int64_t)props.multiProcessorCount * warps_per_sm;

  cuda_err = hipMemsetAsync(permanent_data, 0, sizeof(hipDoubleComplex), stream);
  if (cuda_err != hipSuccess)
    return cuda_err;

  if (h_rows.size() > 0 && minelem != 0)
  {
    size_t new_rows_size = rows_size + 1;
    std::vector<uint64_t> h_new_rows(new_rows_size);
    h_new_rows[0] = 1;
    for (size_t i = 0; i < rows_size; i++)
    {
      h_new_rows[i + 1] = h_rows[i];
    }
    h_new_rows[1 + min_idx] -= 1;

    Matrix<hipDoubleComplex> h_orig_matrix(n, n);
    cuda_err = hipMemcpy(h_orig_matrix.data, A_data, n * n * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    if (cuda_err != hipSuccess)
      return cuda_err;

    Matrix<hipDoubleComplex> mtx_(n + 1, n);
    for (size_t j = 0; j < n; j++)
    {
      mtx_[j] = h_orig_matrix(min_idx, j);
    }
    for (size_t i = 0; i < n; i++)
    {
      for (size_t j = 0; j < n; j++)
      {
        mtx_(i + 1, j) = h_orig_matrix(i, j);
      }
    }

    hipDoubleComplex *d_new_matrix = nullptr;
    uint64_t *d_new_rows = nullptr;
    int *d_n_ary_limits = nullptr;

    cuda_err = hipMalloc(&d_new_matrix, (n + 1) * n * sizeof(hipDoubleComplex));
    if (cuda_err != hipSuccess)
    {
      return cuda_err;
    }
    cuda_err = hipMalloc(&d_new_rows, new_rows_size * sizeof(uint64_t));
    if (cuda_err != hipSuccess)
    {
      hipFree(d_new_matrix);
      return cuda_err;
    }

    cuda_err = hipMemcpyAsync(d_new_matrix, mtx_.data, (n + 1) * n * sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream);
    if (cuda_err != hipSuccess)
    {
      hipFree(d_new_matrix);
      hipFree(d_new_rows);
      return cuda_err;
    }
    cuda_err = hipMemcpyAsync(d_new_rows, h_new_rows.data(), new_rows_size * sizeof(uint64_t), hipMemcpyHostToDevice, stream);
    if (cuda_err != hipSuccess)
    {
      hipFree(d_new_matrix);
      hipFree(d_new_rows);
      return cuda_err;
    }

    Matrix<hipDoubleComplex> modified_m(n + 1, n, d_new_matrix);

    size_t n_ary_size = new_rows_size - 1;
    std::vector<int> h_n_ary_limits(n_ary_size);
    for (size_t i = 0; i < n_ary_size; i++)
    {
      h_n_ary_limits[i] = h_new_rows[i + 1] + 1;
    }

    uint64_t idx_max = h_n_ary_limits[0];
    for (size_t i = 1; i < n_ary_size; i++)
    {
      idx_max *= h_n_ary_limits[i];
    }

    cuda_err = hipMalloc(&d_n_ary_limits, n_ary_size * sizeof(int));
    if (cuda_err != hipSuccess)
    {
      hipFree(d_new_matrix);
      hipFree(d_new_rows);
      return cuda_err;
    }

    cuda_err = hipMemcpyAsync(d_n_ary_limits, h_n_ary_limits.data(), n_ary_size * sizeof(int), hipMemcpyHostToDevice, stream);
    if (cuda_err != hipSuccess)
    {
      hipFree(d_new_matrix);
      hipFree(d_new_rows);
      hipFree(d_n_ary_limits);
      return cuda_err;
    }

    const int block_dim = 256;
    int max_blocks_per_sm;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_blocks_per_sm, PermanentKernelMatrix, block_dim, 0);
    int max_concurrent_blocks_gpu = props.multiProcessorCount * max_blocks_per_sm;
    int min_blocks_for_work = (idx_max == 0) ? 0 : (int)((idx_max + block_dim - 1) / block_dim);
    const int grid_dim = std::min(max_concurrent_blocks_gpu, min_blocks_for_work);

    if (grid_dim > 0)
    {
      PermanentKernelMatrix<<<grid_dim, block_dim, 0, stream>>>(
          modified_m, d_new_rows, new_rows_size,
          cols_data, cols_size,
          d_n_ary_limits, n_ary_size, idx_max,
          host_max_concurrent_warps, sum_rows,
          permanent_data);
      cuda_err = hipGetLastError();
    }

    hipError_t free_err = hipFree(d_new_matrix);
    if (cuda_err == hipSuccess && free_err != hipSuccess)
      cuda_err = free_err;
    free_err = hipFree(d_new_rows);
    if (cuda_err == hipSuccess && free_err != hipSuccess)
      cuda_err = free_err;
    free_err = hipFree(d_n_ary_limits);
    if (cuda_err == hipSuccess && free_err != hipSuccess)
      cuda_err = free_err;

    if (cuda_err != hipSuccess)
      return cuda_err;
  }

  return hipSuccess;
}

ffi::Error PermImpl(hipStream_t stream, ffi::Buffer<ffi::C128> A,
                    ffi::Buffer<ffi::U64> rows,
                    ffi::Buffer<ffi::U64> cols,
                    ffi::ResultBuffer<ffi::C128> permanent)
{
  auto [total_size, n] = get_dims(A);
  size_t rows_size = rows.element_count();
  size_t cols_size = cols.element_count();

  hipDoubleComplex *A_data = reinterpret_cast<hipDoubleComplex *>(A.typed_data());
  uint64_t *rows_data = rows.typed_data();
  uint64_t *cols_data = cols.typed_data();
  hipDoubleComplex *permanent_data = reinterpret_cast<hipDoubleComplex *>(permanent->typed_data());

  hipError_t calc_err = calculatePermanent(stream, A_data, n,
                                            rows_data, rows_size,
                                            cols_data, cols_size,
                                            permanent_data);

  if (calc_err == hipSuccess)
  {
    return ffi::Error::Success();
  }
  else if (calc_err == hipErrorInvalidValue)
  {
    return ffi::Error::InvalidArgument(std::string("Invalid input detected during permanent calculation: ") + hipGetErrorString(calc_err));
  }
  else
  {
    return ffi::Error::Internal(std::string("CUDA error during permanent calculation: ") + hipGetErrorString(calc_err));
  }
}

ffi::Error PermBwdImpl(hipStream_t stream, ffi::Buffer<ffi::C128> res_grad,
                       ffi::Buffer<ffi::C128> A,
                       ffi::Buffer<ffi::U64> rows,
                       ffi::Buffer<ffi::U64> cols,
                       ffi::ResultBuffer<ffi::C128> ct_x)
{
  auto [total_size, n] = get_dims(A);
  if (n == 0)
  {
    hipMemsetAsync(ct_x->typed_data(), 0, total_size * sizeof(hipDoubleComplex), stream);
    return ffi::Error::Success();
  }
  size_t rows_size = rows.element_count();
  size_t cols_size = cols.element_count();
  if (n != rows_size || n != cols_size)
  {
    return ffi::Error::InvalidArgument("Matrix dimension mismatch with row/col vector sizes in PermBwdImpl.");
  }

  std::vector<uint64_t> h_rows(n);
  hipError_t cuda_err = hipMemcpy(h_rows.data(), rows.typed_data(), n * sizeof(uint64_t), hipMemcpyDeviceToHost);
  if (cuda_err != hipSuccess)
    return ffi::Error::Internal(std::string("CUDA memcpy error (h_rows): ") + hipGetErrorString(cuda_err));

  std::vector<uint64_t> h_cols(n);
  cuda_err = hipMemcpy(h_cols.data(), cols.typed_data(), n * sizeof(uint64_t), hipMemcpyDeviceToHost);
  if (cuda_err != hipSuccess)
    return ffi::Error::Internal(std::string("CUDA memcpy error (h_cols): ") + hipGetErrorString(cuda_err));

  hipDoubleComplex *A_data = reinterpret_cast<hipDoubleComplex *>(A.typed_data());
  hipDoubleComplex *ct_x_data = reinterpret_cast<hipDoubleComplex *>(ct_x->typed_data());

  cuda_err = hipMemsetAsync(ct_x_data, 0, n * n * sizeof(hipDoubleComplex), stream);
  if (cuda_err != hipSuccess)
    return ffi::Error::Internal(std::string("CUDA memset error (ct_x): ") + hipGetErrorString(cuda_err));

  for (size_t i = 0; i < n; ++i)
  {
    if (h_rows[i] == 0)
      continue;

    for (size_t j = 0; j < n; ++j)
    {
      if (h_cols[j] == 0)
        continue;

      std::vector<uint64_t> grad_rows_host = h_rows;
      std::vector<uint64_t> grad_cols_host = h_cols;
      grad_rows_host[i] -= 1;
      grad_cols_host[j] -= 1;

      uint64_t *d_grad_rows = nullptr;
      uint64_t *d_grad_cols = nullptr;
      hipDoubleComplex *d_entry_result = nullptr;

      cuda_err = hipMalloc(&d_grad_rows, n * sizeof(uint64_t));
      if (cuda_err != hipSuccess)
      {
        return ffi::Error::Internal(std::string("CUDA malloc error (d_grad_rows): ") + hipGetErrorString(cuda_err));
      }

      cuda_err = hipMalloc(&d_grad_cols, n * sizeof(uint64_t));
      if (cuda_err != hipSuccess)
      {
        hipFree(d_grad_rows);
        return ffi::Error::Internal(std::string("CUDA malloc error (d_grad_cols): ") + hipGetErrorString(cuda_err));
      }

      cuda_err = hipMalloc(&d_entry_result, sizeof(hipDoubleComplex));
      if (cuda_err != hipSuccess)
      {
        hipFree(d_grad_rows);
        hipFree(d_grad_cols);
        return ffi::Error::Internal(std::string("CUDA malloc error (d_entry_result): ") + hipGetErrorString(cuda_err));
      }

      hipMemsetAsync(d_entry_result, 0, sizeof(hipDoubleComplex), stream);
      cuda_err = hipMemcpyAsync(d_grad_rows, grad_rows_host.data(), n * sizeof(uint64_t), hipMemcpyHostToDevice, stream);
      if (cuda_err != hipSuccess)
      {
        hipFree(d_grad_rows);
        hipFree(d_grad_cols);
        hipFree(d_entry_result);
        return ffi::Error::Internal(std::string("CUDA memcpy error (d_grad_rows): ") + hipGetErrorString(cuda_err));
      }

      cuda_err = hipMemcpyAsync(d_grad_cols, grad_cols_host.data(), n * sizeof(uint64_t), hipMemcpyHostToDevice, stream);
      if (cuda_err != hipSuccess)
      {
        hipFree(d_grad_rows);
        hipFree(d_grad_cols);
        hipFree(d_entry_result);
        return ffi::Error::Internal(std::string("CUDA memcpy error (d_grad_cols): ") + hipGetErrorString(cuda_err));
      }

      hipError_t sub_perm_cuda_err = calculatePermanent(stream, A_data, n,
                                                         d_grad_rows, n,
                                                         d_grad_cols, n,
                                                         d_entry_result);

      if (sub_perm_cuda_err != hipSuccess)
      {
        hipFree(d_grad_rows);
        hipFree(d_grad_cols);
        hipFree(d_entry_result);
        if (sub_perm_cuda_err == hipErrorInvalidValue)
        {
          return ffi::Error::InvalidArgument(std::string("Invalid input during sub-permanent calculation: ") + hipGetErrorString(sub_perm_cuda_err));
        }
        else
        {
          return ffi::Error::Internal(std::string("CUDA error during sub-permanent calculation: ") + hipGetErrorString(sub_perm_cuda_err));
        }
      }

      hipDoubleComplex h_entry_result;
      cuda_err = hipMemcpy(&h_entry_result, d_entry_result, sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
      if (cuda_err != hipSuccess)
      {
        hipFree(d_grad_rows);
        hipFree(d_grad_cols);
        hipFree(d_entry_result);
        return ffi::Error::Internal(std::string("CUDA memcpy error (h_entry_result): ") + hipGetErrorString(cuda_err));
      }

      double scale = static_cast<double>(h_rows[i]) * static_cast<double>(h_cols[j]);
      hipDoubleComplex scaled_result = hipCmul(h_entry_result, make_hipDoubleComplex(scale, 0.0));

      cuda_err = hipMemcpyAsync(ct_x_data + i * n + j, &scaled_result, sizeof(hipDoubleComplex), hipMemcpyHostToDevice, stream);
      if (cuda_err != hipSuccess)
      {
        hipFree(d_grad_rows);
        hipFree(d_grad_cols);
        hipFree(d_entry_result);
        return ffi::Error::Internal(std::string("CUDA memcpy error (scaled_result): ") + hipGetErrorString(cuda_err));
      }

      hipFree(d_grad_rows);
      hipFree(d_grad_cols);
      hipFree(d_entry_result);
    }
  }

  cuda_err = hipStreamSynchronize(stream);
  if (cuda_err != hipSuccess)
  {
    return ffi::Error::Internal(std::string("CUDA stream sync error at end of PermBwdImpl: ") + hipGetErrorString(cuda_err));
  }

  return ffi::Error::Success();
}

ffi::Error PermFwdImpl(hipStream_t stream, ffi::Buffer<ffi::C128> A, ffi::Buffer<ffi::U64> rows,
                       ffi::Buffer<ffi::U64> cols,
                       ffi::ResultBuffer<ffi::C128> y,
                       ffi::ResultBuffer<ffi::C128> res)
{
  ffi::Error perm_err = PermImpl(stream, A, rows, cols, y);

  hipError_t cuda_err = hipMemcpyAsync(
      reinterpret_cast<hipDoubleComplex *>(res->typed_data()),
      reinterpret_cast<hipDoubleComplex *>(y->typed_data()),
      sizeof(hipDoubleComplex),
      hipMemcpyDeviceToDevice,
      stream);

  if (cuda_err != hipSuccess)
  {
    return ffi::Error::Internal(std::string("CUDA memcpy error (res): ") + hipGetErrorString(cuda_err));
  }

  hipError_t last_error = hipGetLastError();
  if (last_error != hipSuccess)
  {
    return ffi::Error::Internal(std::string("CUDA error: ") + hipGetErrorString(last_error));
  }
  return ffi::Error::Success();
}
